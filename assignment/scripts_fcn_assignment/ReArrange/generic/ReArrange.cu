#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"

template <typename Dtype>
__global__ void ReArrangeForward(int threads, const Dtype* input_data, Dtype* output_data, int samplenum, int channels, int height, int width ) {

  // int i,h,w,c;
  int imgsize = height * width * channels; 

  CUDA_KERNEL_LOOP(index, threads) 
  {
    int i = index / imgsize;
    int c = (index / (height * width)) % channels; 
    int h = (index / width) % height; 
    int w = index % width; 

    Dtype * now_output = output_data + i * imgsize + (h * width + w) * channels;
    now_output[c] = input_data[i * imgsize + c * height * width + h * width + w];
  }
}


template <typename Dtype>
__global__ void ReArrangeBackward(int threads, const Dtype* gradOutput_data, Dtype* gradInput_data, int samplenum, int channels, int height, int width ) {

  // int i,h,w,c;
  int imgsize = height * width * channels; 

  CUDA_KERNEL_LOOP(index, threads) 
  {
    int i = index / imgsize;
    int c = (index / (height * width)) % channels; 
    int h = (index / width) % height; 
    int w = index % width; 
    
    const Dtype * now_gradOutput = gradOutput_data + i * imgsize + (h * width + w) * channels;
    gradInput_data[i * imgsize + c * height * width + h * width + w]= now_gradOutput[c];
  }

}


static int cunn_ReArrange_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  //THTensor_(resizeAs)(output, input);
  int samplenum = input->size[0];
  int channels = input->size[1];
  int height = input->size[2];
  int width  = input->size[3]; 
  int imgsize = height * width * channels; 

  THCudaTensor_resize2d(state, output, samplenum * height * width, channels);

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);
  float* output_data = THCudaTensor_data(state, output);
  int count = samplenum * imgsize;

  ReArrangeForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data, output_data, samplenum, channels, height, width );

  THCudaTensor_free(state, input);

  return 1;
}

static int cunn_ReArrange_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  
  long samplenum = input->size[0];
  long channels = input->size[1];
  long height = input->size[2];
  long width  = input->size[3]; 
  long imgsize = height * width * channels; 
 

  THCudaTensor_resize4d(state, gradInput, samplenum, channels, height, width);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  float* gradOutput_data = THCudaTensor_data(state, gradOutput);
  float* gradInput_data = THCudaTensor_data(state, gradInput);
  int count = samplenum * imgsize;
  
  ReArrangeBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, gradOutput_data, gradInput_data, samplenum, channels, height, width );

  THCudaTensor_free(state, gradOutput);

  return 1;
}



static const struct luaL_Reg cunn_ReArrange__ [] = {
  {"ReArrange_updateOutput", cunn_ReArrange_updateOutput},
  {"ReArrange_updateGradInput", cunn_ReArrange_updateGradInput},
  {NULL, NULL}
};

void cunn_ReArrange_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_ReArrange__, "nn");
  lua_pop(L,1);
}

