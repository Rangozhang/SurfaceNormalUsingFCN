#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"



template <typename Dtype>
__global__ void ReArrangeBackForward(int threads, const Dtype* input_data, Dtype* output_data, int samplenum, int channels, int height, int width ) {

  // int i,h,w,c;
  int imgsize = height * width * channels; 
  
  CUDA_KERNEL_LOOP(index, threads) 
  {
    int i = index / imgsize;
    int c = (index / (height * width)) % channels; 
    int h = (index / width) % height; 
    int w = index % width; 

    const Dtype* now_input = input_data + i * imgsize + (h * width + w) * channels;
    output_data[i * imgsize + c * height * width + h * width + w] = now_input[c];
    
  }
}


template <typename Dtype>
__global__ void ReArrangeBackBackward(int threads, const Dtype* gradOutput_data, Dtype* gradInput_data, int samplenum, int channels, int height, int width ) {

  // int i,h,w,c;
  int imgsize = height * width * channels; 

  CUDA_KERNEL_LOOP(index, threads) 
  {
    int i = index / imgsize;
    int c = (index / (height * width)) % channels; 
    int h = (index / width) % height; 
    int w = index % width; 

    Dtype * now_gradInput = gradInput_data + i * imgsize + (h * width + w) * channels;
    now_gradInput[c] = gradOutput_data[i * imgsize + c * height * width + h * width + w];
  }
}



static int cunn_ReArrangeBack_updateOutput(lua_State *L)
{

  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");


  int samplenum = luaT_getfieldcheckint(L, 1, "n");
  int height = luaT_getfieldcheckint(L, 1, "h");
  int width = luaT_getfieldcheckint(L, 1, "w");
  int channels = input->size[1];  
  int imgsize = height * width * channels;

  THCudaTensor_resize4d(state, output, samplenum, channels, height, width);

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);
  float* output_data = THCudaTensor_data(state, output);

  int count = samplenum * imgsize;
  ReArrangeBackForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data, output_data, samplenum, channels, height, width );

  THCudaTensor_free(state, input);


  return 1;
}

static int cunn_ReArrangeBack_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  
  int samplenum = luaT_getfieldcheckint(L, 1, "n");
  int height = luaT_getfieldcheckint(L, 1, "h");
  int width = luaT_getfieldcheckint(L, 1, "w");
  int channels = input->size[1];  
  int imgsize = height * width * channels;

  THCudaTensor_resize2d(state, gradInput, samplenum * height * width, channels);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  float* gradOutput_data = THCudaTensor_data(state, gradOutput);
  float* gradInput_data = THCudaTensor_data(state, gradInput);

  int count = samplenum * imgsize;
  ReArrangeBackBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, gradOutput_data, gradInput_data, samplenum, channels, height, width );

  THCudaTensor_free(state, gradOutput);


  return 1;
}



static const struct luaL_Reg cunn_ReArrangeBack__ [] = {
  {"ReArrangeBack_updateOutput", cunn_ReArrangeBack_updateOutput},
  {"ReArrangeBack_updateGradInput", cunn_ReArrangeBack_updateGradInput},
  {NULL, NULL}
};

void cunn_ReArrangeBack_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_ReArrangeBack__, "nn");
  lua_pop(L,1);
}



